#include "hip/hip_runtime.h"
#include "cuda.hpp"

#define BLOCK_SIZE 32

#include <stdio.h>

__global__ void gpu_copy(const int *src, int *dst, const int N, const int M, const int src_pitch, const int dst_pitch)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int block_memory[BLOCK_SIZE][BLOCK_SIZE];
    if (i < N && j < M)
    {
        size_t src_offset = i * src_pitch + j;
        block_memory[threadIdx.y][threadIdx.x] = src[src_offset];
		__syncthreads();
		size_t dst_offset = i/2 * dst_pitch + 2 * j;
		dst[dst_offset] = block_memory[threadIdx.y][threadIdx.x];
		__syncthreads();

		src_offset = (i + 1) * src_pitch + j;
		block_memory[threadIdx.y][threadIdx.x] = src[src_offset];
		__syncthreads();
		dst_offset = i / 2 * dst_pitch + 2 * (j + 1);
		dst[dst_offset] = block_memory[threadIdx.y][threadIdx.x];
		__syncthreads();


		src_offset = (i + 2) * src_pitch + j;
		block_memory[threadIdx.y][threadIdx.x] = src[src_offset];
		__syncthreads();
		dst_offset = (i + 1) / 2 * dst_pitch + 2 * j;
		dst[dst_offset] = block_memory[threadIdx.y][threadIdx.x];


		src_offset = (i + 3) * src_pitch + j;
		block_memory[threadIdx.y][threadIdx.x] = src[src_offset];
		__syncthreads();
		dst_offset = (i + 1) / 2 * dst_pitch + 2 * (j + 1);
		__syncthreads();
    }
}

#include <iomanip>
void gpu(int *src, int *dst, int N, int M)
{
    int *device_orig_matrix = nullptr;
    size_t orig_matrix_memory_pitch = -1;
    hipMallocPitch((void **)&device_orig_matrix, &orig_matrix_memory_pitch, M * sizeof(int), N);

    int *device_dst_matrix = nullptr;
    size_t dst_matrix_memory_pitch = -1;
    hipMallocPitch((void **)&device_dst_matrix, &dst_matrix_memory_pitch, 2 * M * sizeof(int), N);

    hipMemcpy2D(device_orig_matrix, orig_matrix_memory_pitch,
                 src, sizeof(int) * M,
                 sizeof(int) * M, N,
                 hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE),
        grid((M + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    gpu_copy<<<grid, block>>>(device_orig_matrix, device_dst_matrix, N, M, orig_matrix_memory_pitch / 2, dst_matrix_memory_pitch / 2); // WTF ??
    hipDeviceSynchronize();

    hipMemcpy2D(dst, sizeof(int) * M * 2,
                 device_dst_matrix, dst_matrix_memory_pitch,
                 2 * sizeof(int) * M, N,
                 hipMemcpyDeviceToHost);
    hipFree(device_dst_matrix);
    hipFree(device_orig_matrix);
}
