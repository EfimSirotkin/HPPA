#include "hip/hip_runtime.h"
#include "cuda.hpp"

#define BLOCK_SIZE 32

#include <stdio.h>

__global__ void gpu_copy(const short *src, short *dst, const int N, const int M, const int src_pitch, const int dst_pitch)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ short block_memory[BLOCK_SIZE][BLOCK_SIZE];
    if (i < N && j < M)
    {
        size_t src_offset = i * src_pitch + j;
        block_memory[threadIdx.y][threadIdx.x] = src[src_offset];
		__syncthreads();
		size_t dst_offset = i / 2 * dst_pitch + 2 * j;
		dst[dst_offset] = block_memory[threadIdx.y][threadIdx.x];
		__syncthreads();

		src_offset = (i + 1) * src_pitch + j;
		block_memory[threadIdx.y][threadIdx.x] = src[src_offset];
		__syncthreads();
		dst_offset = i / 2 * dst_pitch + 2 * (j + 1);
		dst[dst_offset] = block_memory[threadIdx.y][threadIdx.x];
		__syncthreads();


		src_offset = (i + 2) * src_pitch + j;
		block_memory[threadIdx.y][threadIdx.x] = src[src_offset];
		__syncthreads();
		dst_offset = (i + 1) / 2 * dst_pitch + 2 * j;
		dst[dst_offset] = block_memory[threadIdx.y][threadIdx.x];


		src_offset = (i + 3) * src_pitch + j;
		block_memory[threadIdx.y][threadIdx.x] = src[src_offset];
		__syncthreads();
		dst_offset = (i + 1) / 2 * dst_pitch + 2 * (j + 1);
		__syncthreads();
    }
}

#include <iomanip>
void gpu(short *src, short *dst, int N, int M)
{
    short *device_orig_matrix = nullptr;
    size_t orig_matrix_memory_pitch = -1;
    hipMallocPitch((void **)&device_orig_matrix, &orig_matrix_memory_pitch, M * sizeof(short), N);

    short *device_dst_matrix = nullptr;
    size_t dst_matrix_memory_pitch = -1;
    hipMallocPitch((void **)&device_dst_matrix, &dst_matrix_memory_pitch, 2 * M * sizeof(short), N / 2);

    hipMemcpy2D(device_orig_matrix, orig_matrix_memory_pitch,
                 src, sizeof(short) * M,
                 sizeof(short) * M, N,
                 hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE),
        grid((M + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    gpu_copy<<<grid, block>>>(device_orig_matrix, device_dst_matrix, N, M, orig_matrix_memory_pitch / 2, dst_matrix_memory_pitch / 2); // WTF ??
    hipDeviceSynchronize();

    hipMemcpy2D(dst, sizeof(short) * M * 2,
                 device_dst_matrix, dst_matrix_memory_pitch,
                 2 * sizeof(short) * M, N / 2,
                 hipMemcpyDeviceToHost);
    hipFree(device_dst_matrix);
    hipFree(device_orig_matrix);
}
